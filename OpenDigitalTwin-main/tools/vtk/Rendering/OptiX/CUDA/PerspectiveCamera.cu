#include "hip/hip_runtime.h"
/*
* Copyright (c) 2016, NVIDIA CORPORATION. All rights reserved.
*
* Redistribution and use in source and binary forms, with or without
* modification, are permitted provided that the following conditions
* are met:
*  * Redistributions of source code must retain the above copyright
*    notice, this list of conditions and the following disclaimer.
*  * Redistributions in binary form must reproduce the above copyright
*    notice, this list of conditions and the following disclaimer in the
*    documentation and/or other materials provided with the distribution.
*  * Neither the name of NVIDIA CORPORATION nor the names of its
*    contributors may be used to endorse or promote products derived
*    from this software without specific prior written permission.
*
* THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
* EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
* IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
* PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
* CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
* EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
* PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
* PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
* OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
* (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
* OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/
#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <hip/hip_math_constants.h>

#include "Random.h"
#include "Common.h"

using namespace optix;

rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint2, launch_dim,   rtLaunchDim, );

rtBuffer<uchar4, 2>   frame_buffer;
rtBuffer<float, 2>    depth_buffer;

rtDeclareVariable(rtObject,      top_object, , );
rtDeclareVariable(float3,        pos, , );
rtDeclareVariable(float3,        U, , );
rtDeclareVariable(float3,        V, , );
rtDeclareVariable(float3,        W, , );
rtDeclareVariable(int,           sqrt_num_samples, , );

RT_PROGRAM void PerspectiveCameraRayGen()
{
  size_t2 screen = frame_buffer.size();
  float2 inv_screen = 1.0f/make_float2(screen) * 2.f;
  float2 pixel = (make_float2(launch_index)) * inv_screen - 1.f;

  float2 jitter_scale = inv_screen / (float) sqrt_num_samples;
  int samples_per_pixel = sqrt_num_samples*sqrt_num_samples;

  const float3 ray_origin    = pos;

  float3 result = make_float3(0.0f);

  unsigned int seed = tea<16>(screen.x*launch_index.y+launch_index.x, 0);
  float minDepth = HIP_INF_F;

  do
  {
    int x = samples_per_pixel%sqrt_num_samples;
    int y = samples_per_pixel/sqrt_num_samples;

    float2 jitter = (samples_per_pixel > 1) ? make_float2(x-rnd(seed), y-rnd(seed)) : make_float2(x, y);
    float2 d = pixel + jitter*jitter_scale;
    const float3 ray_direction = normalize(d.x*U + d.y*V + W);

    RadiancePRD prd;
    prd.result = make_float3( 0.0f ); //ray_direction*0.5f + make_float3( 0.5f );
    prd.depth = HIP_INF_F;

    optix::Ray ray = optix::make_Ray(
      ray_origin,
      ray_direction,
      RADIANCE_RAY_TYPE,
      0.001f,
      RT_DEFAULT_MAX
      );

    rtTrace( top_object, ray, prd );
    result += prd.result;
    minDepth = min(prd.depth, minDepth);

  } while(--samples_per_pixel);


  result = result / (sqrt_num_samples * sqrt_num_samples);

  const float3 c = fminf( result, make_float3( 1.0f ) );
  frame_buffer[launch_index] = make_uchar4( c.x*255.99f, c.y*255.99f, c.z*255.99f, 255 );
  depth_buffer[launch_index] = minDepth;
}
