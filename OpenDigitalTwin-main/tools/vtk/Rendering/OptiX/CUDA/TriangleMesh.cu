#include "hip/hip_runtime.h"
/*
* Copyright (c) 2016, NVIDIA CORPORATION. All rights reserved.
*
* Redistribution and use in source and binary forms, with or without
* modification, are permitted provided that the following conditions
* are met:
*  * Redistributions of source code must retain the above copyright
*    notice, this list of conditions and the following disclaimer.
*  * Redistributions in binary form must reproduce the above copyright
*    notice, this list of conditions and the following disclaimer in the
*    documentation and/or other materials provided with the distribution.
*  * Neither the name of NVIDIA CORPORATION nor the names of its
*    contributors may be used to endorse or promote products derived
*    from this software without specific prior written permission.
*
* THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
* EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
* IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
* PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
* CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
* EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
* PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
* PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
* OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
* (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
* OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

#include <optix.h>
#include <optixu/optixu_math_namespace.h>

#include "Common.h"

using namespace optix;

rtBuffer<float3>    vertices;
rtBuffer<float3>    normals;
rtBuffer<int3>      triangles;
rtBuffer<float2>    texcoords;
rtBuffer<float4>    vertexcolors;

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );

rtDeclareVariable( IntersectionRecord, irec, attribute irec, );

RT_PROGRAM void TriangleMeshIntersection( int prim_idx )
{
  const int3 v_idx = triangles[prim_idx];

  const float3 p0 = vertices[ v_idx.x ];
  const float3 p1 = vertices[ v_idx.y ];
  const float3 p2 = vertices[ v_idx.z ];

  // Intersect ray with triangle
  float3 n;
  float  t, beta, gamma;
  if( intersect_triangle( ray, p0, p1, p2, n, t, beta, gamma ) )
  {
    if(  rtPotentialIntersection( t ) )
    {
      irec.PrimIdx = prim_idx;

      irec.Ng = normalize( n );
      if( normals.size() == 0 )
      {
        irec.N = irec.Ng;
      }
      else
      {
        const float3 n0 = normals[ v_idx.x ];
        const float3 n1 = normals[ v_idx.y ];
        const float3 n2 = normals[ v_idx.z ];
        irec.N = normalize( n1*beta + n2*gamma + n0*(1.0f-beta-gamma) );
      }

      if(texcoords.size() == 0)
      {
        irec.TexCoord = make_float2(0.0);
      }
      else
      {
        const float2 t0 = texcoords[ v_idx.x ];
        const float2 t1 = texcoords[ v_idx.y ];
        const float2 t2 = texcoords[ v_idx.z ];
        irec.TexCoord = t1*beta + t2*gamma + t0*(1.0f-beta-gamma);
      }

      if(vertexcolors.size() == 0)
      {
        irec.VertexColor = make_float3(1.0f, 1.0f, 1.0f);
      }
      else
      {
        const float4 c0 = vertexcolors[ v_idx.x ];
        const float4 c1 = vertexcolors[ v_idx.y ];
        const float4 c2 = vertexcolors[ v_idx.z ];
        float4 temp = c1*beta + c2*gamma + c0*(1.0f-beta-gamma);
        irec.VertexColor = make_float3(temp.x, temp.y, temp.z);
      }

      rtReportIntersection( 0 );
    }
  }
}


RT_PROGRAM void TriangleMeshBoundingBox( int prim_idx, float result[6] )
{
    const int3 v_idx = triangles[prim_idx];

    const float3 v0   = vertices[ v_idx.x ];
    const float3 v1   = vertices[ v_idx.y ];
    const float3 v2   = vertices[ v_idx.z ];
    const float  area = length(cross(v1-v0, v2-v0));

    optix::Aabb* aabb = (optix::Aabb*)result;

    if( area > 0.0f && !isinf(area) )
    {
      aabb->m_min = fminf( fminf( v0, v1), v2 );
      aabb->m_max = fmaxf( fmaxf( v0, v1), v2 );
    }
    else
    {
      aabb->invalidate();
    }
}
