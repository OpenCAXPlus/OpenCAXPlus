#include "hip/hip_runtime.h"
/*
* Copyright (c) 2016, NVIDIA CORPORATION. All rights reserved.
*
* Redistribution and use in source and binary forms, with or without
* modification, are permitted provided that the following conditions
* are met:
*  * Redistributions of source code must retain the above copyright
*    notice, this list of conditions and the following disclaimer.
*  * Redistributions in binary form must reproduce the above copyright
*    notice, this list of conditions and the following disclaimer in the
*    documentation and/or other materials provided with the distribution.
*  * Neither the name of NVIDIA CORPORATION nor the names of its
*    contributors may be used to endorse or promote products derived
*    from this software without specific prior written permission.
*
* THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
* EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
* IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
* PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
* CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
* EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
* PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
* PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
* OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
* (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
* OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

#include <optix.h>
#include <optixu/optixu_math_namespace.h>

#include "Common.h"

using namespace optix;

rtBuffer<float4>    vertices;
rtBuffer<int2>      lines;
rtBuffer<float2>    texcoords;
rtBuffer<float4>    vertexcolors;

rtDeclareVariable( optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable( IntersectionRecord, irec, attribute irec, );

rtDeclareVariable( float, cylinder_radius, , );

template<bool use_robust_method>
static __device__
void intersect_cylinder(int prim_idx)
{
  int2 vertexIdxs = lines[prim_idx];
  float4 posAndRadius0 = vertices[vertexIdxs.x];
  float4 posAndRadius1 = vertices[vertexIdxs.y];

  float3 pos0 = make_float3(posAndRadius0.x, posAndRadius0.y, posAndRadius0.z);
  float3 pos1 = make_float3(posAndRadius1.x, posAndRadius1.y, posAndRadius1.z);

  const float radius = (posAndRadius0.w + posAndRadius1.w) * 0.5 * cylinder_radius;

  float3 O = ray.origin - pos0;
  float3 D = ray.direction; //Assumed to be of length 1
  float3 C = pos1 - pos0;

  float dotCC = dot(C,C);
  float dotOO = dot(O,O);
  float dotDC = dot(D,C);
  float dotOC = dot(O,C);
  float dotOD = dot(O,D);
  float dotCCInv = (dotCC == 0.0f ? 1.0f : 1.0f/dotCC);

  float dotDCdotCCInv = dotDC*dotCCInv;
  float dotOCdotCCInv = dotOC*dotCCInv;

  float a = 1-dotDC*dotDCdotCCInv;
  float b = dotOD - dotOC*dotDCdotCCInv;
  float c = dotOO - dotOC*dotOCdotCCInv - radius*radius;

  //A factor 2 falls away in b, and in the second term of discriminant
  float disc = b*b-a*c;

  if(disc > 0.0f)
  {
    float sdisc = sqrtf(disc);
    float root1 = (-b - sdisc)/a;

    //bool do_refine = false;

    float root11 = 0.0f;

    bool check_second = true;

    float dC1 = dotOCdotCCInv + dotDCdotCCInv * root1;

    if( dC1 >= 0.0f && dC1 <= 1.0f && rtPotentialIntersection( root1 + root11 ) )
    {
      float3 pC = dC1 * C;
      float3 pR = O + root1 * D;

      irec.PrimIdx = prim_idx;
      irec.N = irec.Ng = (pR - pC)/radius;

      irec.TexCoord = (1-dC1)*texcoords[vertexIdxs.x] + dC1*texcoords[vertexIdxs.y];


      if(vertexcolors.size() == 0)
      {
        irec.VertexColor = make_float3(1.0f, 1.0f, 1.0f);
      }
      else
      {
        float4 vertexColor = (1-dC1)*vertexcolors[vertexIdxs.x] + dC1*vertexcolors[vertexIdxs.y];
        irec.VertexColor = make_float3(vertexColor.x, vertexColor.y, vertexColor.z);
      }

      if(rtReportIntersection(0))
      {
        check_second = false;
      }
    }

    if(check_second)
    {
      float root2 = (-b + sdisc) / a; //+ (do_refine ? root1 : 0);

      float dC2 = dotOCdotCCInv + dotDCdotCCInv * root2;

      if( dC2 >= 0.0f && dC2 <= 1.0f && rtPotentialIntersection( root2 ) )
      {
        float3 pC = dC2 * C;
        float3 pR = O + root2 * D;

        irec.PrimIdx = prim_idx;
        irec.N = irec.Ng = (pR - pC)/radius;

        irec.TexCoord = (1-dC2)*texcoords[vertexIdxs.x] + dC2*texcoords[vertexIdxs.y];


        if(vertexcolors.size() == 0)
        {
          irec.VertexColor = make_float3(1.0f, 1.0f, 1.0f);
        }
        else
        {
          float4 vertexColor = (1-dC2)*vertexcolors[vertexIdxs.x] + dC2*vertexcolors[vertexIdxs.y];

          irec.VertexColor = make_float3(vertexColor.x, vertexColor.y, vertexColor.z);
        }

        rtReportIntersection(0);
      }
    }
  }
}

RT_PROGRAM void CylinderIntersect( int prim_idx )
{
  intersect_cylinder<false>(prim_idx);
}

RT_PROGRAM void CylinderIntersect_robust(int prim_idx)
{
  intersect_cylinder<true>(prim_idx);
}

RT_PROGRAM void CylinderBounds (int prim_idx, float result[6])
{
  int2 vertexIdxs = lines[prim_idx];
  float4 posAndRadius0 = vertices[vertexIdxs.x];
  float4 posAndRadius1 = vertices[vertexIdxs.y];

  float3 pos0 = make_float3(posAndRadius0.x, posAndRadius0.y, posAndRadius0.z);
  float3 pos1 = make_float3(posAndRadius1.x, posAndRadius1.y, posAndRadius1.z);

  const float radius = (posAndRadius0.w + posAndRadius1.w) * 0.5 * cylinder_radius;

  optix::Aabb* aabb = (optix::Aabb*)result;

  if( radius > 0.0f  && !isinf(radius) )
  {
    aabb->m_min = fminf(pos0, pos1) - radius;
    aabb->m_max = fmaxf(pos0, pos1) + radius;
  }
  else
  {
    aabb->invalidate();
  }
}
