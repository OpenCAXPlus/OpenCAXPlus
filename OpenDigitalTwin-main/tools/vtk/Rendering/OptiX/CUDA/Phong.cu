#include "hip/hip_runtime.h"
/*
* Copyright (c) 2016, NVIDIA CORPORATION. All rights reserved.
*
* Redistribution and use in source and binary forms, with or without
* modification, are permitted provided that the following conditions
* are met:
*  * Redistributions of source code must retain the above copyright
*    notice, this list of conditions and the following disclaimer.
*  * Redistributions in binary form must reproduce the above copyright
*    notice, this list of conditions and the following disclaimer in the
*    documentation and/or other materials provided with the distribution.
*  * Neither the name of NVIDIA CORPORATION nor the names of its
*    contributors may be used to endorse or promote products derived
*    from this software without specific prior written permission.
*
* THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
* EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
* IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
* PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
* CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
* EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
* PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
* PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
* OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
* (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
* OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <hip/hip_math_constants.h>

#include "Random.h"
#include "Common.h"
#include "Light.h"

#include <stdio.h>

using namespace optix;

rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint2, launch_dim,   rtLaunchDim, );

rtDeclareVariable(rtObject, top_object, , );

rtBuffer<vtkopt::Light> lights;
rtBuffer<float3> cellcolors;

rtDeclareVariable( RadiancePRD, prd,   rtPayload, );
rtDeclareVariable( optix::Ray,  ray,   rtCurrentRay, );
rtDeclareVariable( float,       t_hit, rtIntersectionDistance, );

rtDeclareVariable( float3, Ks, , );
rtDeclareVariable( float3, Kd, , );
rtDeclareVariable( float,  Ns, , );

rtDeclareVariable( float3, bg_color, , );
rtDeclareVariable( int, shadows_enabled, , );
rtDeclareVariable( int, texture_enabled, , );
rtDeclareVariable( int, vertexcolors_enabled, , );
rtDeclareVariable( int, cellcolors_enabled, , );
rtDeclareVariable( int, num_ambient_samples, , );
rtDeclareVariable( float, ambient_occlusion_dist, , );
rtDeclareVariable( float, occlusion_epsilon, , );

rtDeclareVariable( IntersectionRecord, irec, attribute irec, );

rtTextureSampler<uchar4, 2, hipReadModeNormalizedFloat> colorTexture;

static __device__
float3 cartesianCoords(float phi, float sinTheta, float cosTheta)
{
  float sinPhi, cosPhi;
  sincos(phi, &sinPhi, &cosPhi);
  return make_float3(cosPhi * sinTheta,
                     sinPhi * sinTheta,
                     cosTheta);
}

static __device__
float3 cosineSampleHemisphere(float2 s_uni)
{
  float cosTheta = sqrt(s_uni.y);
  float sinTheta = sqrt(1.0f - s_uni.y);
  float phi = 2 * HIP_PI_F * s_uni.x;
  return cartesianCoords(phi, sinTheta, cosTheta);
}

static __device__
void buildFrame( float3 N, float3* mat )
{
  float3 tan0 = make_float3( 0.0f, N.z, -N.y );
  float3 tan1 = make_float3( -N.z, 0.0f, N.x );
  float3 tan = normalize( abs(N.x) < abs(N.y) ? tan0 : tan1 );
  float3 bitan = cross( N, tan );
  mat[0] = tan;
  mat[1] = bitan;
  mat[2] = N;
}

RT_PROGRAM void LambertianClosestHit()
{
  const float3 N = faceforward( irec.N, -ray.direction, irec.Ng );
  const float3 P  = ray.origin + t_hit * ray.direction;
  //const float3 Kd = make_float3( 0.7f, 0.7f, 0.7f );

  float3 Kdiffuse = Kd;
  if( texture_enabled )
  {
    float4 temp = tex2D( colorTexture, irec.TexCoord.x, irec.TexCoord.y );
    Kdiffuse = make_float3(temp.x, temp.y, temp.z);
  }
  else if( vertexcolors_enabled )
  {
    Kdiffuse = irec.VertexColor;
  }
  else if( cellcolors_enabled )
  {
    Kdiffuse = cellcolors[irec.PrimIdx];
  }

  // light loop
  float3 color = make_float3( 0.0f );
  const int num_lights = lights.size();
  for( int i =0; i < num_lights; ++i )
  {
    const vtkopt::Light light = lights[i];
    float3 L;
    float  Ldist;
    float3  Lcolor;
    if( light.type == vtkopt::Light::DIRECTIONAL )
    {
      L = -light.dir;
      Ldist = 1e8f;
      Lcolor = light.color;
    }
    else
    {
      Ldist = optix::length( light.pos - P );
      L = ( light.pos-P ) / Ldist;
      Lcolor = light.color/(Ldist*Ldist);
    }

    const float N_dot_L = optix::dot( L, N );

    float3 light_attenuation = make_float3( 1.0f );
    if( N_dot_L > 0.0f )
    {
      //
      // Calculation occlusion
      //
      if( shadows_enabled )
      {
        OcclusionPRD shadow_prd;
        shadow_prd.occlusion = make_float3( 1.0f );

        optix::Ray shadow_ray =
          optix::make_Ray(
            P,
            L,
            OCCLUSION_RAY_TYPE,
            occlusion_epsilon,
            Ldist );
        rtTrace( top_object, shadow_ray, shadow_prd );

        light_attenuation = shadow_prd.occlusion;
      }

      //
      // Calculate local lighting
      //
      if( fmaxf(light_attenuation) > 0.0f )
      {
        //const float3 H = optix::normalize( L - ray.direction );
        //const float  N_dot_H = optix::dot( N, H );
        const float3 R = optix::reflect( ray.direction, N );
        //clamp, as normal can be slightly inaccurate, causing aliasing with large Ns
        const float L_dot_R = fminf(fmaxf( optix::dot( L, R ), 0.0f), 1.0f);
        color += ( Kdiffuse*N_dot_L + Ks*powf( L_dot_R, Ns ) ) * Lcolor * light_attenuation;
      }
    }
  }

  if( num_ambient_samples > 0 )
  {
    float3 localToWorld[3];
    buildFrame(N, localToWorld);

    int Pi0 = *((int*)(&P.x));
    int Pi1 = *((int*)(&P.y));
    int Pi2 = *((int*)(&P.z));

    unsigned int seed = tea<16>( Pi0^Pi1^Pi2, 0 );

    float3 attenuation = make_float3(0.0f);
    for( int ambSampleIdx = 0; ambSampleIdx < num_ambient_samples; ++ambSampleIdx )
    {
      float2 uniSamples = make_float2( rnd(seed), rnd(seed) );
      float3 ambDir = cosineSampleHemisphere( uniSamples );

      ambDir = localToWorld[0]*ambDir.x + localToWorld[1]*ambDir.y + localToWorld[2]*ambDir.z;

      if( dot( ambDir, N ) < 0.05f )
      {
        continue;
      }

      OcclusionPRD ambient_prd;
      ambient_prd.occlusion = make_float3( 1.0f );

      optix::Ray ambient_ray =
        optix::make_Ray(
          P,
          ambDir,
          OCCLUSION_RAY_TYPE,
          occlusion_epsilon,
          ambient_occlusion_dist );
      rtTrace( top_object, ambient_ray, ambient_prd );

      attenuation += ambient_prd.occlusion;
    }
    attenuation = attenuation/num_ambient_samples;

    color *= attenuation;
  }

  prd.result = color;
  prd.depth = t_hit;
}

rtDeclareVariable( OcclusionPRD, shadow_prd, rtPayload, );

RT_PROGRAM void LambertianAnyHit()
{
  shadow_prd.occlusion = make_float3( 0.0f );
}

RT_PROGRAM void Miss()
{
  prd.result = bg_color;
}
