#include "hip/hip_runtime.h"
/*
* Copyright (c) 2016, NVIDIA CORPORATION. All rights reserved.
*
* Redistribution and use in source and binary forms, with or without
* modification, are permitted provided that the following conditions
* are met:
*  * Redistributions of source code must retain the above copyright
*    notice, this list of conditions and the following disclaimer.
*  * Redistributions in binary form must reproduce the above copyright
*    notice, this list of conditions and the following disclaimer in the
*    documentation and/or other materials provided with the distribution.
*  * Neither the name of NVIDIA CORPORATION nor the names of its
*    contributors may be used to endorse or promote products derived
*    from this software without specific prior written permission.
*
* THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
* EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
* IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
* PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
* CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
* EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
* PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
* PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
* OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
* (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
* OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

#include <optix.h>
#include <optixu/optixu_math_namespace.h>

#include "Common.h"

using namespace optix;

rtBuffer<float4>    spheres;
rtBuffer<float2>    texcoords;
rtBuffer<float4>    vertexcolors;

rtDeclareVariable( optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable( IntersectionRecord, irec, attribute irec, );

rtDeclareVariable( float, sphere_radius, , );

template<bool use_robust_method>
static __device__
void intersect_sphere(int prim_idx)
{
  float4 sphereAndRadius = spheres[prim_idx];

  const float3 center = make_float3( sphereAndRadius.x, sphereAndRadius.y, sphereAndRadius.z );
  const float radius = sphereAndRadius.w * sphere_radius;

  float3 O = ray.origin - center;
  float3 D = ray.direction;

  float b = dot(O, D);
  float c = dot(O, O)-radius*radius;
  float b2 = b*b;
  if(b2 > c)
  {
    float sdisc = sqrtf(b2 - c);
    float root1 = (-b - sdisc);

    bool do_refine = false;

    float root11 = 0.0f;

    if(use_robust_method && /*fabsf(root1) > 10.f * radius*/c > b2*0.999f )
    {
      do_refine = true;
    }

    if(do_refine)
    {
      // refine root1
      float3 O1 = O + root1 * ray.direction;
      b = dot(O1, D);
      c = dot(O1, O1) - radius*radius;
      float disc = b*b - c;

      if(disc > 0.0f)
      {
        sdisc = sqrtf(disc);
        root11 = (-b - sdisc);
      }
    }

    bool check_second = true;
    if( rtPotentialIntersection( root1 + root11 ) )
    {
      irec.PrimIdx = prim_idx;
      irec.N = irec.Ng = (O + (root1 + root11)*D)/radius;

      irec.TexCoord = texcoords[prim_idx];

      if(vertexcolors.size() == 0)
      {
        irec.VertexColor = make_float3(1.0f, 1.0f, 1.0f);
      }
      else
      {
        float4 vertexColor = vertexcolors[prim_idx];
        irec.VertexColor = make_float3(vertexColor.x, vertexColor.y, vertexColor.z);
      }

      if(rtReportIntersection(0))
      {
        check_second = false;
      }
    }

    if(check_second)
    {
      float root2 = (-b + sdisc) + (do_refine ? root1 : 0);

      if( rtPotentialIntersection( root2 ) )
      {
        irec.PrimIdx = prim_idx;
        irec.N = irec.Ng = (O + root2*D)/radius;
        irec.TexCoord = texcoords[prim_idx];

        if(vertexcolors.size() == 0)
        {
          irec.VertexColor = make_float3(1.0f, 1.0f, 1.0f);
        }
        else
        {
          float4 vertexColor = vertexcolors[prim_idx];
          irec.VertexColor = make_float3(vertexColor.x, vertexColor.y, vertexColor.z);
        }

        rtReportIntersection(0);
      }
    }
  }
}

RT_PROGRAM void SphereIntersect( int prim_idx )
{
  intersect_sphere<true>(prim_idx);
}

RT_PROGRAM void SphereIntersect_robust(int prim_idx)
{
  intersect_sphere<true>(prim_idx);
}

RT_PROGRAM void SphereBounds (int prim_idx, float result[6])
{
  float4 sphereAndRadius = spheres[prim_idx];

  const float3 center = make_float3( sphereAndRadius.x, sphereAndRadius.y, sphereAndRadius.z );
  const float3 radius = make_float3( sphereAndRadius.w * sphere_radius );

  optix::Aabb* aabb = (optix::Aabb*)result;

  if( radius.x > 0.0f  && !isinf(radius.x) )
  {
    aabb->m_min = center - radius;
    aabb->m_max = center + radius;
  }
  else
  {
    aabb->invalidate();
  }
}
